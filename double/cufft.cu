#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <sys/time.h>
#include "common.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
using namespace std;
void cufft1d(double *signal, long int row, Complex *r_signal, int batch){
	long int bytes=batch*row*sizeof(hipfftDoubleComplex);
	hipfftDoubleComplex* h_signal=new hipfftDoubleComplex[row*batch];
	for(long int i=0; i<row*batch; i++){
		h_signal[i].x=signal[i];
		h_signal[i].y=0.;
	}
	hipfftDoubleComplex* d_signal, *o_signal;
	hipMalloc((void**)&d_signal, bytes);
	hipMalloc((void**)&o_signal, bytes);
	hipMemcpy(d_signal, h_signal, bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	int dim_arry[3]={row, 1, 1};
	//hipfftPlan1d(&plan, row, HIPFFT_C2C, 1);
	hipfftPlanMany(&plan, 1, dim_arry, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, batch);
	//warm up
	hipfftExecZ2Z(plan, d_signal, o_signal, HIPFFT_FORWARD);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for(int i=0; i<COUNT; i++)
		hipfftExecZ2Z(plan, d_signal, d_signal, HIPFFT_FORWARD);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime=0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime *=1.e-3;
	float gflops=COUNT*batch*row*5*log(row)/log(2)/elapsedTime*1.e-9;
	cout<<row<<", "<<elapsedTime/COUNT<<" s/cufft1d "<<gflops<<" gflops"<<endl;
	hipMemcpy(r_signal,(Complex*)d_signal, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	delete[] h_signal;
	hipFree(d_signal);
}
void cufft2d(double *signal, int row, int col, Complex *r_signal, int batch){
	int bytes=batch*row*col*sizeof(hipfftDoubleComplex);
	hipfftDoubleComplex* h_signal=new hipfftDoubleComplex[row*col*batch];
	for(unsigned int i=0; i<row*col*batch; i++){
		h_signal[i].x=signal[i];
		h_signal[i].y=0.;
	}
	hipfftDoubleComplex* d_signal, *o_signal;
	hipMalloc((void**)&d_signal, bytes);
	hipMalloc((void**)&o_signal, bytes);
	hipMemcpy(d_signal, h_signal, bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	int dim_arry[3]={row, col, 1};
	hipfftPlanMany(&plan, 2, dim_arry, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, batch);
	//hipfftPlan2d(&plan, row, col, HIPFFT_C2C);
	//warm up
	//hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for(int i=0; i<COUNT; i++)
		hipfftExecZ2Z(plan, d_signal, d_signal, HIPFFT_FORWARD);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime=0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime *=1.e-3;
	float gflops = batch*COUNT*5*row*col*log(row*col)/elapsedTime*1.e-9;
	cout<<"row="<<row<<" col="<<col<<", "<<elapsedTime/COUNT<<" s/cufft2d "<<gflops<<" gflops"<<endl;
	hipMemcpy(r_signal,(Complex*)d_signal, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	delete[] h_signal;
	hipFree(d_signal);
}
void cufft3d(double *signal, int row, int col, int nz, Complex *r_signal, int batch){
	int bytes=batch*row*col*nz*sizeof(hipfftDoubleComplex);
	hipfftDoubleComplex* h_signal=new hipfftDoubleComplex[row*col*nz*batch];
	for(unsigned int i=0; i<row*col*nz*batch; i++){
		h_signal[i].x=signal[i];
		h_signal[i].y=0.;
	}
	hipfftDoubleComplex* d_signal, *o_signal;
	hipMalloc((void**)&d_signal, bytes);
	hipMalloc((void**)&o_signal, bytes);
	hipMemcpy(d_signal, h_signal, bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	int dim_arry[3]={row, col, nz};
	hipfftPlanMany(&plan, 3, dim_arry, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, batch);
	//hipfftPlan3d(&plan, row, col, nz, HIPFFT_C2C);
	//warm up
	//hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for(int i=0; i<COUNT; i++)
		hipfftExecZ2Z(plan, d_signal, d_signal, HIPFFT_FORWARD);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime=0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime *=1.e-3;
	float gflops = batch*COUNT*5*row*col*nz*log(row*col*nz)/elapsedTime*1.e-9;
	cout<<"row="<<row<<" col="<<col<<" nz="<<nz<<", "<<elapsedTime/COUNT<<" s/cufft3d "<<gflops<<" gflops"<<endl;
	hipMemcpy(r_signal,(Complex*)d_signal, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	delete[] h_signal;
	hipFree(d_signal);
}
