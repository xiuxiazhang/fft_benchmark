#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <sys/time.h>
#include "common.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
using namespace std;
void cufft1d(float *signal, long int row, Complex *r_signal, int batch){
	long int bytes=batch*row*sizeof(hipfftComplex);
	hipfftComplex* h_signal=new hipfftComplex[row*batch];
	for(long int i=0; i<row*batch; i++){
		h_signal[i].x=signal[i];
		h_signal[i].y=0.;
	}
	hipfftComplex* d_signal, *o_signal;
	hipMalloc((void**)&d_signal, bytes);
	hipMalloc((void**)&o_signal, bytes);
	hipMemcpy(d_signal, h_signal, bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	int dim_arry[3]={row, 1, 1};
	hipfftPlan1d(&plan, row, HIPFFT_C2C, 1);
	hipfftPlanMany(&plan, 1, dim_arry, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch);
	//warm up
	hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for(int i=0; i<COUNT; i++)
		hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime=0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime *=1.e-3;
	float gflops=COUNT*batch*row*5*log(row)/log(2)/elapsedTime*1.e-9;
	cout<<row<<", "<<elapsedTime/COUNT<<" s/cufft1d "<<gflops<<" gflops"<<endl;
	hipMemcpy(r_signal,(Complex*)o_signal, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	delete[] h_signal;
	hipFree(d_signal);
	hipFree(o_signal);
}
void cufft2d(float *signal, int row, int col, Complex *r_signal, int batch){
	int bytes=batch*row*col*sizeof(hipfftComplex);
	hipfftComplex* h_signal=new hipfftComplex[row*col*batch];
	for(unsigned int i=0; i<row*col*batch; i++){
		h_signal[i].x=signal[i];
		h_signal[i].y=0.;
	}
	hipfftComplex* d_signal, *o_signal;
	hipMalloc((void**)&d_signal, bytes);
	hipMalloc((void**)&o_signal, bytes);
	hipMemcpy(d_signal, h_signal, bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	int dim_arry[3]={row, col, 1};
	hipfftPlanMany(&plan, 2, dim_arry, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch);
	//hipfftPlan2d(&plan, row, col, HIPFFT_C2C);
	//warm up
	//hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for(int i=0; i<COUNT; i++)
		hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime=0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime *=1.e-3;
	float gflops = batch*COUNT*5*row*col*log(row*col)/log(2)/elapsedTime*1.e-9;
	cout<<"row="<<row<<" col="<<col<<", "<<elapsedTime/COUNT<<" s/cufft2d "<<gflops<<" gflops"<<endl;
	hipMemcpy(r_signal,(Complex*)o_signal, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	delete[] h_signal;
	hipFree(d_signal);
	hipFree(o_signal);
}
void cufft3d(float *signal, int row, int col, int nz, Complex *r_signal, int batch){
	int bytes=batch*row*col*nz*sizeof(hipfftComplex);
	hipfftComplex* h_signal=new hipfftComplex[row*col*nz*batch];
	for(unsigned int i=0; i<row*col*nz*batch; i++){
		h_signal[i].x=signal[i];
		h_signal[i].y=0.;
	}
	hipfftComplex* d_signal, *o_signal;
	hipMalloc((void**)&d_signal, bytes);
	hipMalloc((void**)&o_signal, bytes);
	hipMemcpy(d_signal, h_signal, bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	int dim_arry[3]={row, col, nz};
	hipfftPlanMany(&plan, 3, dim_arry, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch);
	//hipfftPlan3d(&plan, row, col, nz, HIPFFT_C2C);
	//warm up
	//hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for(int i=0; i<COUNT; i++)
		hipfftExecC2C(plan, d_signal, o_signal, HIPFFT_FORWARD);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime=0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime *=1.e-3;
	float gflops = batch*COUNT*5*row*col*nz*log(row*col*nz)/log(2)/elapsedTime*1.e-9;
	cout<<"row="<<row<<" col="<<col<<" nz="<<nz<<", "<<elapsedTime/COUNT<<" s/cufft2d "<<gflops<<" gflops"<<endl;
	hipMemcpy(r_signal,(Complex*)o_signal, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	delete[] h_signal;
	hipFree(d_signal);
	hipFree(o_signal);
}
